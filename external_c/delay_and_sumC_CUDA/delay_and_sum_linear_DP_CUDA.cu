
#include <hip/hip_runtime.h>
//*****************************************************
// FILE CONTAINS 
//      GPU DEVICE KERNEL
//      EXTERNAL C CODE (GPU calls)
//*****************************************************

//=========================================================================================================
//=========================================================================================================
// (GPU Cuda Kernel) Double Precision Delay and Sum Algorithm, using linear interpolation 
//=========================================================================================================
//=========================================================================================================
__global__ void c_delay_and_sum_amplitudes_linear_Kernel_DP( const double* scanlines,  const unsigned int* tx,  const unsigned int* rx,  const double* lookup_times_tx,  const double* lookup_times_rx,
                 const double* amplitudes_tx,  const double* amplitudes_rx, double invdt, double t0, double fillvalue,double* result,
                int numpoints,int numsamples,int numelements, int numscanlines)
{
    int point = threadIdx.x + blockIdx.x * blockDim.x;
    if (point < numpoints)
    {
        int scan,lookup_index,t_ind,r_ind,set_val,pixie,point2,lookup_index1,set_val1;
        double tot_real1,tot_imag1,amp_corr,lookup_time,frac1;
        tot_real1=0.0;
        tot_imag1=0.0;
        pixie=numelements*point;
        for (scan=0;scan<numscanlines;scan++)
        {  
            t_ind=pixie+tx[scan];
            r_ind=pixie+rx[scan];
            lookup_time = (lookup_times_tx[t_ind] + lookup_times_rx[r_ind] - t0)* invdt;
            lookup_index = (int)lookup_time;
            frac1 = lookup_time - lookup_index;
            lookup_index1=lookup_index+1;
            if (lookup_index < 0)  
            {
            }
            else if (lookup_index1 >= numsamples)
            {
            }
            else
            {
                amp_corr= amplitudes_tx[t_ind] * amplitudes_rx[r_ind];
                set_val=scan*numsamples+lookup_index;
                set_val=set_val*2;
                set_val1=set_val+2;
                tot_real1 += amp_corr * scanlines[set_val]+frac1*(scanlines[set_val1]-scanlines[set_val]); 
                tot_imag1 += amp_corr * scanlines[set_val+1]+frac1*(scanlines[set_val1+1]-scanlines[set_val+1]); 
            }  
        }  
        point2=point*2;
        result[point2]=tot_real1;
        result[point2+1]=tot_imag1;  
    }    
}  

//=========================================================================================================
//=========================================================================================================
// DOUBLE PRECISION EXTERNAL C CODE (LINEAR INTERPOLATION)
//=========================================================================================================
//=========================================================================================================
int c_delayAndSum_Algorithm_linear_DP_CUDA(
    const double* scanlines,                     //INPUT double complex array as [:,:,2] 2 doubles [numscanlines x numsamples x 2]
    const unsigned int* tx,                      //INPUT unsigned int array [:] with size [numscanlines] transmitter
    const unsigned int* rx,                      //INPUT unsigned int array [:] with size [numscanlines] receiver
    const double* lookup_times_tx,               //INPUT double array [:,:] with size [numpoints * numelements] Flight time to transmitter from each point
    const double* lookup_times_rx,               //INPUT double array [:,:] with size [numpoints * numelements] Flight time to receiver from each point
    const double* amplitudes_tx,                 //INPUT double array [:,:] with size [numpoints * numelements] Transmitter Amplitude Array  
    const double* amplitudes_rx,                 //INPUT double array [:,:] with size [numpoints * numelements] Receiver Amplitude Array  
    double invdt,                                //INPUT double inverse time step 1/dt
    double t0,                                   //INPUT double initial time instance
    double fillvalue,                            //INPUT double fillvalue (if calculated lookup time outside sample range)
    double* result,                              //OUTPUT double array [:,2] with size [numpoints x 2 ] 2 doubles representing double complex.
    int numpoints,                               //INPUT int number of points in TFM
    int numsamples,                              //INPUT int number of time samples in scanlines
    int numelements,                             //INPUT int number of elements in Probe Array
    int numscanlines,                            //INPUT int number of scanlines - which is 0.5*(numelements*numelements+numelements) for HMC, numelements*numelements for FMC
    int NTHREADS,                                //INPUT int Number of threads (for GPU)
    int txUpdate,                                //INPUT int txUpdate = 0 (don't update Transmitter focal law information on GPU) 1 (Update this information) 
    int rxUpdate,                                //INPUT int rxUpdate = 0 (don't update Receiver focal law information on GPU) 1 (Update this information)
    int ExpDataUpdate,                           //INPUT int ExpDataUpdate = 0 (don't update scanlines information on GPU) 1 (Update this information)
    int finaliseOpt)                             //INPUT int finaliseOpt <0 Don't run finalise Stage, finaliseOpt = 0 finalise after processing, finaliseOpt=1 finalise and immediate return.

{
    /*********************************************************************************\
     Calculation of Linear Interpolation DELAY and SUM Algorithm 
     ^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^
     Algorithm uses GPU
      
     When using GPU, Four stages can be considered
     1. Optional Initialisation (Memory Allocation on GPU) if not yet allocated.
     2. Transfer of information to GPU (altered values only)
     3. Calculation usign KERNEL, transfer of OUTPUT data from GPU to Host
     
     4. Optional Cleanup of GPU (Deallocation of GPU Memory)  
        STAGE 4 can be run independently. Only Stage 4 is run if finaliseOpt = 1
    
     
        
    \********************************************************************************/

    //Local
    int Nscan=numscanlines*numsamples;
    int Nlook=numpoints*numelements;
    //Input Data Pointers    
    static double *scanlines_gpu;
    static unsigned int *tx_gpu;
    static unsigned int *rx_gpu;
    static double *lookup_times_tx_gpu;
    static double *lookup_times_rx_gpu;
    static double *amplitudes_tx_gpu;
    static double *amplitudes_rx_gpu;
    //Output Data Pointers
    static double *result_gpu; 
    //Returned
    static int Marker=0;

    // Initialise GPU Device if required
    // Marker = 0 (GPU Memory not yet allocated)
    if (Marker < 1) //Not yet allocated
    {
        // Input
        hipMalloc(&scanlines_gpu, Nscan*2*sizeof(double)); 
        hipMalloc(&tx_gpu, numscanlines*sizeof(unsigned int)); 
        hipMalloc(&rx_gpu, numscanlines*sizeof(unsigned int)); 
        hipMalloc(&lookup_times_tx_gpu, Nlook*sizeof(double));
        hipMalloc(&lookup_times_rx_gpu, Nlook*sizeof(double));
        hipMalloc(&amplitudes_tx_gpu, Nlook*sizeof(double));
        hipMalloc(&amplitudes_rx_gpu, Nlook*sizeof(double));
        // Output
        hipMalloc(&result_gpu, numpoints*2*sizeof(double));
        //Mark that initialisation has occurred
        //======================================================================
        Marker=1;        
        // Ensure that data is transferred after initialisation
        txUpdate=1;
        rxUpdate=1;
        ExpDataUpdate=1;
    }
    
    // OPTIONAL USER FORCED CLEANUP (AND IMMEDIATE RETURN)
    if (finaliseOpt == 1)
    {
            hipFree(scanlines_gpu);
            hipFree(tx_gpu);
            hipFree(rx_gpu);
            hipFree(lookup_times_tx_gpu);
            hipFree(lookup_times_rx_gpu);
            hipFree(amplitudes_tx_gpu);
            hipFree(amplitudes_rx_gpu);
            hipFree(result_gpu);
            hipDeviceReset();
            Marker=0;
            return Marker;
    }
    
    // Copy any new data
    //==========================================================================    
    
    if (ExpDataUpdate == 1)
    {
        //Transfer ExpData related data to GPU 
        //======================================================================
        hipMemcpy(scanlines_gpu, scanlines, Nscan*2*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(tx_gpu, tx, numscanlines*sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(rx_gpu, rx, numscanlines*sizeof(unsigned int), hipMemcpyHostToDevice);  
    }
    
    if (txUpdate == 1)
    {
        //Transfer Transmitter related data to GPU 
        //======================================================================
        hipMemcpy(lookup_times_tx_gpu, lookup_times_tx, Nlook*sizeof(double), hipMemcpyHostToDevice); 
        hipMemcpy(amplitudes_tx_gpu, amplitudes_rx, Nlook*sizeof(double), hipMemcpyHostToDevice);  
    }
    if (rxUpdate == 1)
    {
        //Transfer Receiver related data to GPU 
        //======================================================================
        hipMemcpy(lookup_times_rx_gpu, lookup_times_rx, Nlook*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(amplitudes_rx_gpu, amplitudes_rx, Nlook*sizeof(double), hipMemcpyHostToDevice);         
    }

    // Calculate Block Size for GPU
    int blockSize,minGridSize,gridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, c_delay_and_sum_amplitudes_linear_Kernel_DP, 0, numpoints); 
    // Round up according to array size 
    gridSize = (numpoints + blockSize - 1) / blockSize; 

    // Perform Kernel Operations
    //=========================================================================
    c_delay_and_sum_amplitudes_linear_Kernel_DP<<<gridSize, blockSize>>>(scanlines_gpu,tx_gpu,rx_gpu,lookup_times_tx_gpu,lookup_times_rx_gpu,
                 amplitudes_tx_gpu,amplitudes_rx_gpu, invdt, t0, fillvalue,result_gpu,
                numpoints,numsamples,numelements, numscanlines);


    // Return Data from GPU
    //=========================================================================
    hipMemcpy(result, result_gpu, numpoints*2*sizeof(double), hipMemcpyDeviceToHost);
    
    Marker++; //Counts Up (for reporting purposes, can see if initialisation in current run (initialisation =2, else >2)
    
    // OPTIONAL USER FORCED CLEANUP
    if (finaliseOpt == 0)
    {
            hipFree(scanlines_gpu);
            hipFree(tx_gpu);
            hipFree(rx_gpu);
            hipFree(lookup_times_tx_gpu);
            hipFree(lookup_times_rx_gpu);
            hipFree(amplitudes_tx_gpu);
            hipFree(amplitudes_rx_gpu);
            hipFree(result_gpu);
            hipDeviceReset();
            Marker=0;
    }

    return Marker;
}

